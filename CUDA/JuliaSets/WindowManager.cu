#include "WindowManager.h"
#include "FrameManager.h"
#include "Frame.h"
#include "Config.h"
#include "Tools.h"

#include "SDL.h"

SDL_Surface* window;
Frame* mainFrame;
Frame* secFrame;

// Updates the display
void updateWindow() {

    unsigned char* pixels = updateFrame(mainFrame);

    for(int y = 0; y < mainFrame->heightPixels; y++)
        for(int x = 0; x < mainFrame->widthPixels; x++)
            setPixel(window, x, y, SDL_MapRGB(
                        window->format,
                        0,
                        pixels[y*(mainFrame->widthPixels)+x],
                        0));

    //Updates the screen
    SDL_Flip(window);

    hipFree(pixels);
}

void openWindow() {

    SDL_Init(SDL_INIT_VIDEO);
    window = SDL_SetVideoMode(DEFAULT_WIDTH, DEFAULT_HEIGHT, 32, SDL_HWSURFACE);
    SDL_WM_SetCaption("Mandelbrot: Left click to zoom, Right click to unzoom", NULL);

    mainFrame = initFrame(DEFAULT_WIDTH/2, DEFAULT_HEIGHT, 0);
    secFrame = initFrame(DEFAULT_WIDTH/2, DEFAULT_HEIGHT, 2);
}

void closeWindow() {
    destroyFrame(mainFrame);
    destroyFrame(secFrame);
    SDL_Quit();
}


//Returns 0 if the user closed the window and 1 otherwise
//Updates the screen values (zooming/resizing)
int updateEvents() {

    int res = -1;
    while(res == -1) {

        SDL_Event event;
        while (SDL_PollEvent(&event)) {

            switch(event.type) {

                case SDL_QUIT:
                    res = 0;
                    break;

                case SDL_MOUSEBUTTONDOWN:
                    int mainFocused = event.button.x <= mainFrame->widthPixels;
                    zoom(
                        mainFocused ? mainFrame : secFrame,
                        event.button.x - (mainFocused ? 0 : mainFrame->widthPixels),
                        event.button.y,
                        event.button.button ==
                            SDL_BUTTON_LEFT ? ZOOM_STRENGTH : 1.0/ZOOM_STRENGTH
                    );

                    res = 1;
                    break;
            }
        }

        int mouseX;
        int mouseY;
        SDL_GetMouseState(&mouseX, &mouseY);

        if(mouseX <= mainFrame->widthPixels) {

            unsigned char* pixels = updateFrame(secFrame);

            for(int y = 0; y < secFrame->heightPixels; y++)
                for(int x = 0; x < secFrame->widthPixels; x++)
                    setPixel(window, x+(mainFrame->widthPixels), y, SDL_MapRGB(
                                window->format,
                                0,
                                 pixels[y*(secFrame->widthPixels)+x],
                                0));

            secFrame->additionnalData[0] = mainFrame->topLeftX + (float)mouseX/(mainFrame->widthPixels) * mainFrame->width;
            secFrame->additionnalData[1] = mainFrame->topLeftY - (float)mouseY/(mainFrame->heightPixels) * mainFrame->height;

            SDL_Flip(window);

            hipFree(pixels);
        }
    }

    return res;
}
