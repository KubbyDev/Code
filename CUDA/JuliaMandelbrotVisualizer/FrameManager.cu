#include "FrameManager.h"
#include "Mandelbrot.h"
#include "Frame.h"
#include "Config.h"

Frame* initFrame(int width, int height, int posx, int posy, int additionnalDataCount) {

    Frame* frame;
    hipMallocManaged(&frame, sizeof(Frame));

    frame->xPixels = posx;
    frame->yPixels = posy;
    frame->widthPixels = width;
    frame->heightPixels = height;
    frame->width = 2;
    frame->height = (frame->width) * (float)height/width;
    frame->topLeftX = -1;
    frame->topLeftY = (frame->height)/2;

    frame->additionnalDataCount = additionnalDataCount;
    if(additionnalDataCount > 0) {
        float* data;
        hipMallocManaged(&data, sizeof(float)*additionnalDataCount);
        frame->additionnalData = data;
    }

    return frame;
}

unsigned char* updateFrame(Frame* frame) {

    unsigned char* pixels;
    hipMallocManaged(&pixels, sizeof(unsigned char)*(frame->widthPixels)*(frame->heightPixels));

    calculatePixels<<<NB_BLOCKS, NB_THREADS>>>(frame, pixels);
    hipDeviceSynchronize();

    return pixels;
}

//Zooms. newCenter is the position of the new center of the window in the screen plane
void zoom(Frame* frame, int newCenterX, int newCenterY, float zoomFactor) {

    zoomFactor = 1/zoomFactor; //Inverts the zoomFactor to make it zoom when it is >1

    frame->topLeftX +=
        ((float)newCenterX/(frame->widthPixels))*(frame->width) //Position of the new center in real plane
        - (frame->width)/2 *zoomFactor;                         //Position of the new top left corner in real plane
    frame->topLeftY -=
        ((float)newCenterY/(frame->heightPixels))*(frame->height)
        - (frame->height)/2 *zoomFactor;

    frame->width *= zoomFactor;
    frame->height *= zoomFactor;
}

void destroyFrame(Frame* frame) {
    if(frame->additionnalDataCount > 0)
        hipFree(frame->additionnalData);
    hipFree(frame);
}
