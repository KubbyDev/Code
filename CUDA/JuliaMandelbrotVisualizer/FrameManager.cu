#include "FrameManager.h"
#include "ImageRenderer.h"
#include "Frame.h"
#include "Config.h"
#include "Color.h"

Frame* initFrame(int width, int height, int posx, int posy, int additionnalDataCount) {

    Frame* frame;
    hipMallocManaged(&frame, sizeof(Frame));

    frame->xPixels = posx;
    frame->yPixels = posy;
    frame->widthPixels = width;
    frame->heightPixels = height;
    frame->width = 2;
    frame->height = (frame->width) * (float)height/width;
    frame->topLeftX = -1;
    frame->topLeftY = (frame->height)/2;

    frame->additionnalDataCount = additionnalDataCount;
    if(additionnalDataCount > 0) {
        float* data;
        hipMallocManaged(&data, sizeof(float)*additionnalDataCount);
        frame->additionnalData = data;
    }

    hipMallocManaged(&(frame->pixels), sizeof(Color)*(frame->widthPixels)*(frame->heightPixels));

    return frame;
}

Color* updateFrame(Frame* frame) {

    calculatePixels<<<NB_BLOCKS, NB_THREADS>>>(frame);
    hipDeviceSynchronize();

    return frame->pixels;
}

//Zooms. newCenter is the position of the new center of the window in the screen plane
void zoom(Frame* frame, int newCenterX, int newCenterY, float zoomFactor) {

    zoomFactor = 1/zoomFactor; //Inverts the zoomFactor to make it zoom when it is >1

    frame->topLeftX +=
        ((float)newCenterX/(frame->widthPixels))*(frame->width) //Position of the new center in real plane
        - (frame->width)/2 *zoomFactor;                         //Position of the new top left corner in real plane
    frame->topLeftY -=
        ((float)newCenterY/(frame->heightPixels))*(frame->height)
        - (frame->height)/2 *zoomFactor;

    frame->width *= zoomFactor;
    frame->height *= zoomFactor;
}

void resize(Frame* frame, int newWidth, int newHeight, int newX, int newY) {

    frame->widthPixels = newWidth;
    frame->heightPixels = newHeight;

    frame->xPixels = newX;
    frame->yPixels = newY;

    frame->height = (frame->width*newHeight)/newWidth;

    hipFree(frame->pixels);
    hipMallocManaged(&(frame->pixels), sizeof(Color)*newWidth*newHeight);
}

void destroyFrame(Frame* frame) {
    if(frame->additionnalDataCount > 0)
        hipFree(frame->additionnalData);
    hipFree(frame->pixels);
    hipFree(frame);
}
