#include "hip/hip_runtime.h"
#include "ImageRenderer.h"
#include "Frame.h"
#include "Config.h"
#include "Color.h"
#include "ColorationFunctions.h"

__device__
// Returns the index at which z^2 + c becomes more than 2, divided by the threshold
// 1 means the threshold is reached but the term's modulus is still less than 2
// 0 means the first term's modulus is aleady more than 2
// The threshold can be seen as the clearness of the drawing
float getConvergenceSpeed(float zx, float zy, float cx, float cy, int threshold) {

    int i = 0;
    while (zx*zx + zy*zy < 4 && i < threshold) {

	    float tmp = zx*zx - zy*zy + cx;
	    zy = 2*zx*zy + cy;
	    zx = tmp;

	    i++;
    }

    return (float)i/threshold;
}

__global__
// Calculates all the pixels of the given frame on the GPU
// The results are values between 0 and 1 in resPixels
void calculatePixels(Frame* frame) {

    Color* resPixels = frame->pixels;

    int threadId = threadIdx.x + blockIdx.x*blockDim.x;
    //(wP, hP) is the width/height of the screen (number of pixels)
    //(tlX, tlY) is the position of the top left corner of the screen in the real plane
    //(w, h) is the size of the screen in the real plane's coordinates
    int wP = frame->widthPixels;
    int hP = frame->heightPixels;
    float tlX = frame->topLeftX;
    float tlY = frame->topLeftY;
    float w = frame->width;
    float h = frame->height;

    int totalPixels = wP*hP;
    // +1 makes sure all pixels are calculated. Maybe a thread will do less
    // work but thats not problematic
    int pixelPerThread = totalPixels/(NB_THREADS*NB_BLOCKS) +1;

    //Calculates the new pixel values
    for(int i = threadId*pixelPerThread; i < pixelPerThread*(threadId+1); i++) {

        // Avoids calculating pixels that will not be displayed
        if(i >= totalPixels)
            break;

        float pixelValue;
        if(frame->additionnalDataCount >= 2) {
            // If we are calculating a Julia pixel
            pixelValue = getConvergenceSpeed(
                    tlX + (float)(i%wP)/wP * w,
                    tlY - (float)i/(wP*hP) * h,
                    frame->additionnalData[0],
                    frame->additionnalData[1],
                    JULIA_ITERATIONS_THRESHOLD
            );
        }
        else {
            // If we are calculating a Mandelbrot pixel
            pixelValue = getConvergenceSpeed(
                    0,
                    0,
                    tlX + (float)(i%wP)/wP * w,
                    tlY - (float)i/(wP*hP) * h,
                    MANDELBROT_ITERATIONS_THRESHOLD
            );
        }

        //Updates the pixel color
        COLORATION_FUNCTION(pixelValue, resPixels+i);
    }
}
