#include "hip/hip_runtime.h"
#include "Mandelbrot.h"
#include "Frame.h"
#include "Config.h"

//(x0, y0) is the position of the pixel in the real plane (not the screen plane)
__device__
inline float getMandelbrotValue(float x0, float y0) {

    //This value can be seen as the clearness of the drawing
    int max_i = MANDELBROT_ITERATIONS_THRESHOLD;

    float x = 0, y = 0;
    int i = 0;
    while (x*x + y*y < 4 && i < max_i) {

	    float x_temp = x*x - y*y + x0;
	    y = 2*x*y + y0;
	    x = x_temp;

	    i++;
    }

    return (float)i/max_i;
}

//(x0, y0) is the position of the pixel in the complex plane
//(cx, cy) is the c in z^2 + c
__device__
inline float getJuliaValue(float x0, float y0, float cx, float cy) {

    //This value can be seen as the clearness of the drawing
    int max_i = JULIA_ITERATIONS_THRESHOLD;

    float x = x0, y = y0;
    int i = 0;
    while (x*x + y*y < 4 && i < max_i) {

	    float x_temp = x*x - y*y + cx;
	    y = 2*x*y + cy;
	    x = x_temp;

	    i++;
    }

    return (float)i/max_i;
}

__global__
void calculatePixels(Frame* frame, unsigned char* resPixels) {

    int threadId = threadIdx.x + blockIdx.x*blockDim.x;
    //(wP, hP) is the width/height of the screen (number of pixels)
    //(tlX, tlY) is the position of the top left corner of the screen in the real plane
    //(w, h) is the size of the screen in the real plane's coordinates
    int wP = frame->widthPixels;
    int hP = frame->heightPixels;
    float tlX = frame->topLeftX;
    float tlY = frame->topLeftY;
    float w = frame->width;
    float h = frame->height;
    int pixelPerThread = (wP*hP)/(NB_THREADS*NB_BLOCKS);

    //Calculates the new pixel values
    for(int i = threadId*pixelPerThread; i < pixelPerThread*(threadId+1); i++) {

        float pixelValue;
        if(frame->additionnalDataCount >= 2) {
            pixelValue = getJuliaValue(
                tlX + (float)(i%wP)/wP * w,
                tlY - (float)i/(wP*hP) * h,
                frame->additionnalData[0],
                frame->additionnalData[1]
            );
        }
        else {
            pixelValue = getMandelbrotValue(
                tlX + (float)(i%wP)/wP * w,
                tlY - (float)i/(wP*hP) * h
            );
        }

        //Updates the pixel color
        resPixels[i] = 255*pixelValue;
    }
}
