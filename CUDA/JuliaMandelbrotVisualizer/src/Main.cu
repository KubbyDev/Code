#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "WindowManager.h"

int main() {

    openWindow();

    // Initialises the fps counter
    unsigned long frames = 0;
    time_t seconds = time(NULL);

    while (updateWindow()) { // Returns 0 if the user closed the program
        
        frames++;
        
        // Each second, displays the number of frames calculated during that second
        if (seconds != time(NULL)) {
            printf("Fps: %lu\n", frames);
            frames = 0;
        }
        seconds = time(NULL);
    }

    closeWindow();

    return 0;
}
